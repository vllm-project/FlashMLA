#include "hip/hip_runtime.h"
// SM90-specific file - only compile for SM90+ architectures
#include "fwd.h"
#include <stdexcept>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900) && (__CUDA_ARCH__ < 1000) || !defined(__CUDA_ARCH__)

#include <hip/hip_math_constants.h>
#include <cute/tensor.hpp>
#include <cutlass/cluster_launch.hpp>
#include <hip/hip_cooperative_groups.h>
#include <cutlass/arch/reg_reconfig.h>
#include <cutlass/arch/arch.h>

#include "utils.h"
#include "helpers.h"

namespace sm90 {

using namespace cute;

constexpr int D_Q = 576;
constexpr int D_K = 576;
constexpr int D_V = 512;

constexpr int B_H = 64;
constexpr int B_TOPK = 64;    // TopK block size
constexpr int NUM_THREADS = 128*3;
static constexpr float MAX_INIT_VAL = -1e30;    // We use this number as the initial value for mi (max logits)

template<int NUM_TILES>
using SmemLayoutQTiles = decltype(coalesce(tile_to_shape(
    GMMA::Layout_K_SW128_Atom<bf16>{},
    Shape<Int<B_H>, Int<64*NUM_TILES>>{},
    Step<_1, _2>{}
), Shape<_1, _1>{}));

template<int NUM_TILES>
using SmemLayoutOTiles = decltype(coalesce(tile_to_shape(
    GMMA::Layout_K_SW128_Atom<bf16>{},
    Shape<Int<B_H>, Int<64*NUM_TILES>>{},
    Step<_1, _2>{}
), Shape<_1, _1>{}));

template<int NUM_TILES>
using SmemLayoutKTiles = decltype(coalesce(tile_to_shape(
    GMMA::Layout_SW128_Atom<bf16, GMMA::Major::K>{},
    Shape<Int<B_TOPK>, Int<64*NUM_TILES>>{},
    Step<_1, _2>{}
), Shape<_1, _1>{}));

template<int NUM_TILES>
using SmemLayoutKTilesTransposed = decltype(composition(
	SmemLayoutKTiles<NUM_TILES>{},
	Layout<Shape<Int<64*NUM_TILES>, Int<B_TOPK>>, Stride<Int<B_TOPK>, _1>>{}
));

using SmemLayoutQ = SmemLayoutQTiles<9>;
using SmemLayoutO = SmemLayoutOTiles<8>;
using SmemLayoutK = SmemLayoutKTiles<9>;
using SmemLayoutV = SmemLayoutKTilesTransposed<8>;
using SmemLayoutHalfV = SmemLayoutKTilesTransposed<4>;

using SmemLayoutS = decltype(coalesce(tile_to_shape(
    GMMA::Layout_K_SW128_Atom<bf16>{},
    Shape<Int<B_H>, Int<B_TOPK>>{}
), Shape<_1, _1>{}));

struct SharedMemoryPlan {
    union {
        array_aligned<bf16, cosize_v<SmemLayoutQ>> q;
        array_aligned<bf16, cosize_v<SmemLayoutO>> o;
    } q_o;
    array_aligned<bf16, cosize_v<SmemLayoutK>> k[2];
    array_aligned<bf16, cosize_v<SmemLayoutS>> s;

    bool is_kv_valid[2][B_TOPK];
    float2 sM[32];
    float2 sL[64];   // For reduction across WG0/1 in epilogue
    float final_max_logits[64], final_lse[64];
    transac_bar_t bar_q, bar_k0_free[2], bar_k0_ready[2], bar_k1_free[2], bar_k1_ready[2], bar_is_kv_valid_ready;
};

using TiledMMA_QK = decltype(make_tiled_mma(
    GMMA::MMA_64x64x16_F32BF16BF16_SS<GMMA::Major::K, GMMA::Major::K>{},
    Layout<Shape<_1, _1, _1>>{}
));

using TiledMMA_PV_LocalP = decltype(make_tiled_mma(
    GMMA::MMA_64x256x16_F32BF16BF16_RS<GMMA::Major::K, GMMA::Major::MN>{},
    Layout<Shape<_1, _1, _1>>{}
));

using TiledMMA_PV_RemoteP = decltype(make_tiled_mma(
    GMMA::MMA_64x256x16_F32BF16BF16_SS<GMMA::Major::K, GMMA::Major::MN>{},
    Layout<Shape<_1, _1, _1>>{}
));

template<
    typename Shape_Q, typename TMA_Q
>
struct TmaParams {
    Shape_Q shape_Q; TMA_Q tma_Q;
    CUtensorMap tensor_map_O;
};

enum NamedBarriers : uint32_t {
    wg0_bunch_0_ready = 0,
    wg1_bunch_0_ready = 1,
    wg0_s0_ready = 2,
    wg1_s1_ready = 3,
    sL_ready = 4,
    warpgroup0_sync = 5,
    warpgroup1_sync = 6
};

// Save rPb (64x64, bfloat16) to sP using the stmatrix instruction
template<
    typename Tensor0,
    typename Tensor1
>
__forceinline__ __device__ void save_rS_to_sS(
    Tensor0 const &rPb,
    Tensor1 const &sP,
    int idx_in_warpgroup
) {
    auto r2s_copy = make_tiled_copy_C(
        Copy_Atom<SM90_U32x4_STSM_N, bf16>{},
        TiledMMA_QK{}
    );
    ThrCopy thr_copy = r2s_copy.get_slice(idx_in_warpgroup);
    Tensor thr_copy_rPb = thr_copy.retile_S(rPb);
    Tensor thr_copy_sP = thr_copy.partition_D(sP);
    cute::copy(r2s_copy, thr_copy_rPb, thr_copy_sP);
}


template<typename TmaParams>
__global__ void __launch_bounds__(NUM_THREADS, 1, 1)
sparse_attn_fwd_kernel(__grid_constant__ const SparsePrefillParams params, __grid_constant__ const TmaParams tma_params) {
    // NOTE This kernel uses a similar schedule to Flash MLA - 0422. For a detailed explanation, please refer to https://github.com/deepseek-ai/FlashMLA/blob/main/docs/20250422-new-kernel-deep-dive.md
#if IS_SM90
    const int q_h_idx = blockIdx.x % (params.h_q/B_H);
    const int s_q_idx = blockIdx.x / (params.h_q/B_H);
    const int warpgroup_idx = cutlass::canonical_warp_group_idx();
    const int warp_idx = cutlass::canonical_warp_idx_sync();
    const int idx_in_warpgroup = threadIdx.x % 128;

    // Define shared tensors
    extern __shared__ char wksp_buf[];
    SharedMemoryPlan &plan = *reinterpret_cast<SharedMemoryPlan*>(wksp_buf);
    Tensor sQ = make_tensor(make_smem_ptr(plan.q_o.q.data()), SmemLayoutQ{});
    Tensor sO = make_tensor(make_smem_ptr(plan.q_o.o.data()), SmemLayoutO{});
    Tensor sS0 = make_tensor(make_smem_ptr(plan.k[0].data()+64*512), SmemLayoutS{});    // Overlap with sK0's RoPE part
    Tensor sS1 = make_tensor(make_smem_ptr(plan.s.data()), SmemLayoutS{});

    if (warp_idx == 0 && elect_one_sync()) {
        // Prefetch TMA descriptors
        cute::prefetch_tma_descriptor(tma_params.tma_Q.get_tma_descriptor());
        cute::prefetch_tma_descriptor(&tma_params.tensor_map_O);

        // Initialize barriers
        plan.bar_q.init(1);
        CUTE_UNROLL
        for (int i = 0; i < 2; ++i) {
            plan.bar_k0_free[i].init(128);
            plan.bar_k0_ready[i].init(128);
            plan.bar_k1_free[i].init(128);
            plan.bar_k1_ready[i].init(128);
        }
        plan.bar_is_kv_valid_ready.init(16);
        fence_barrier_init();
    }

    __syncthreads();
    
    const int num_topk_blocks = params.topk / B_TOPK;
    if (warpgroup_idx == 0 || warpgroup_idx == 1) {
        cutlass::arch::warpgroup_reg_alloc<216>();

        if (warp_idx == 0 && elect_one_sync()) {
            // Load Q
            Tensor gQ = flat_divide(
                tma_params.tma_Q.get_tma_tensor(tma_params.shape_Q)(_, _, s_q_idx),
                Tile<Int<B_H>, Int<D_Q>>{}
            )(_, _, q_h_idx, _0{});
            launch_tma_copy(tma_params.tma_Q, gQ, sQ, plan.bar_q, TMA::CacheHintSm90::EVICT_FIRST);
            plan.bar_q.arrive_and_expect_tx(B_H*D_Q*sizeof(bf16));
        }

        float rM[2] = {MAX_INIT_VAL, MAX_INIT_VAL}; // Meaning: the `max_logits` used for O / rL calculation
        float rL[2] = {0.0f, 0.0f};
        Tensor rO = partition_fragment_C(TiledMMA_PV_LocalP{}, Shape<Int<B_H>, Int<D_V/2>>{});
        Tensor rP = partition_fragment_C(TiledMMA_QK{}, Shape<Int<B_H>, Int<B_TOPK>>{});
        Tensor rS = make_tensor<bf16>(partition_shape_A(TiledMMA_PV_LocalP{}, Shape<Int<B_H>, Int<B_TOPK>>{}));
        cute::fill(rO, 0.0f);
        
        // Wait for Q
        plan.bar_q.wait(0);

        bool cur_bar_wait_phase = 0;
        
        struct Warpgroup0 {};
        struct Warpgroup1 {};

        auto qkt_gemm_one_tile = [&](auto warpgroup_idx, int tile_idx, bool clear_accum) {
            constexpr bool IS_WG1 = std::is_same_v<decltype(warpgroup_idx), Warpgroup1>;
            TiledMMA tiled_mma_QK = TiledMMA_QK{};
            Tensor sQ_tile = flat_divide(sQ, Tile<Int<B_H>, Int<64>>{})(_, _, _0{}, tile_idx);
            Tensor sK_tile = make_tensor(make_smem_ptr(plan.k[(int)IS_WG1].data() + tile_idx*B_TOPK*64), SmemLayoutKTiles<1>{});
            gemm_ss(clear_accum, tiled_mma_QK, sQ_tile, sK_tile, rP, idx_in_warpgroup);
        };

        auto mask_rP = [&](auto warpgroup_idx) {
            constexpr bool IS_WG1 = std::is_same_v<decltype(warpgroup_idx), Warpgroup1>;
            plan.bar_is_kv_valid_ready.wait(cur_bar_wait_phase);
            CUTE_UNROLL
            for (int row_idx = 0; row_idx < 2; ++row_idx) {
                CUTE_UNROLL
                for (int i = row_idx*2; i < size(rP); i += 4) {
                    int col = 8*(i/4) + (idx_in_warpgroup%4)*2;
                    if (!plan.is_kv_valid[IS_WG1][col]) rP(i) = -INFINITY;
                    if (!plan.is_kv_valid[IS_WG1][col+1]) rP(i+1) = -INFINITY;
                }
            }
        };

        auto online_softmax_and_rescale_o = [&](auto warpgroup_idx) {
            plan.bar_is_kv_valid_ready.wait(cur_bar_wait_phase);
            constexpr bool IS_WG1 = std::is_same_v<decltype(warpgroup_idx), Warpgroup1>;
            const float scale = params.sm_scale_div_log2;
            float r_sM[2];
            if constexpr (IS_WG1) {
                *(float2*)r_sM = plan.sM[idx_in_warpgroup/4];
            }
            float new_maxs[2];
            CUTE_UNROLL
            for (int row_idx = 0; row_idx < 2; ++row_idx) {
                // Get rowwise max
                float cur_max = -INFINITY;
                CUTE_UNROLL
                for (int i = row_idx*2; i < size(rP); i += 4) {
                    cur_max = max(cur_max, max(rP(i), rP(i+1)));
                }
                cur_max = max(cur_max, __shfl_xor_sync(0xffffffff, cur_max, 1));
                cur_max = max(cur_max, __shfl_xor_sync(0xffffffff, cur_max, 2));
                cur_max *= scale;

                // Get new max and scale
                // For WG1, old_max comes from sM (written by WG0); for WG0, old_max comes from rM (read by WG0 from sM in the last round)
                new_maxs[row_idx] = max(IS_WG1 ? r_sM[row_idx] : rM[row_idx], cur_max);

                // Scale O
                float scale_for_o = exp2f(rM[row_idx]-new_maxs[row_idx]);
                CUTE_UNROLL
                for (int i = row_idx*2; i < size(rO); i += 4) {
                    rO(i) *= scale_for_o;
                    rO(i+1) *= scale_for_o;
                }

                // Get rS
                float cur_sum = 0;
                CUTE_UNROLL
                for (int i = row_idx*2; i < size(rP); i += 4) {
                    rP(i) = exp2f(rP(i)*scale - new_maxs[row_idx]);
                    rP(i+1) = exp2f(rP(i+1)*scale - new_maxs[row_idx]);
                    rS(i) = (bf16)rP(i);
                    rS(i+1) = (bf16)rP(i+1);
                    cur_sum += rP(i) + rP(i+1);
                }
                rL[row_idx] = rL[row_idx]*scale_for_o + cur_sum;
            }
            __syncwarp();
            if (idx_in_warpgroup%4 == 0) {
                plan.sM[idx_in_warpgroup/4] = *(float2*)new_maxs;
            }
            rM[0] = new_maxs[0];
            rM[1] = new_maxs[1];
        };

        auto reduce_L = [&]() {
            // Reduce L
            // For example, thread 0 reduces with thread 1, 2, and 3, as well as thread 128, 129, 130, and 131
            rL[0] += __shfl_xor_sync(0xffffffff, rL[0], 1);
            rL[0] += __shfl_xor_sync(0xffffffff, rL[0], 2);
            rL[1] += __shfl_xor_sync(0xffffffff, rL[1], 1);
            rL[1] += __shfl_xor_sync(0xffffffff, rL[1], 2);
            if (idx_in_warpgroup%4 == 0)
                plan.sL[threadIdx.x/4] = *(float2*)(rL);
            NamedBarrier::arrive_and_wait(256, NamedBarriers::sL_ready);
            float2 peer_L = plan.sL[(threadIdx.x/4)^32];
            rL[0] += peer_L.x;
            rL[1] += peer_L.y;
        };

        auto store_O = [&]() {
            float scale_factors[2];
            CUTE_UNROLL
            for (int i = 0; i < 2; ++i)
                scale_factors[i] = rL[i] == 0.0f ? 1.0f : 1.0f / rL[i];

            Tensor sO = make_tensor(make_smem_ptr(plan.q_o.o.data() + warpgroup_idx*B_H*(D_V/2)), SmemLayoutOTiles<4>{});
            bf16* stsm_addrs[4];
            int stsm_row = (idx_in_warpgroup/32)*16 + (idx_in_warpgroup%16);
            CUTE_UNROLL
            for (int i = 0; i < 64/16; ++i) {
                stsm_addrs[i] = &sO(stsm_row, (idx_in_warpgroup%32/16*8) + 16*i);
            }
            bool s2g_pred = warp_idx%4 == 0 && elect_one_sync();

            warpgroup_wait<0>();
            CUTE_UNROLL
            for (int tile_idx = 0; tile_idx < (D_V/2)/64; tile_idx += 1) {
                // Convert
                constexpr int NUM_ELEMS_EACH_TILE = B_H*64 / 128;   // 64: tile size, 128: warpgroup size
                bf16 cur_rOb[NUM_ELEMS_EACH_TILE];
                CUTE_UNROLL
                for (int i = 0; i < NUM_ELEMS_EACH_TILE; ++i) {
                    cur_rOb[i] = (bf16)(rO(tile_idx*NUM_ELEMS_EACH_TILE + i) * scale_factors[i%4>=2]);
                }
                // R -> S
                CUTE_UNROLL
                for (int i = 0; i < 64/16; ++i) {
                    SM90_U32x4_STSM_N::copy(
                        *reinterpret_cast<uint32_t*>(cur_rOb + i*8 + 0),
                        *reinterpret_cast<uint32_t*>(cur_rOb + i*8 + 2),
                        *reinterpret_cast<uint32_t*>(cur_rOb + i*8 + 4),
                        *reinterpret_cast<uint32_t*>(cur_rOb + i*8 + 6),
                        *reinterpret_cast<uint128_t*>(stsm_addrs[i] + tile_idx*(B_H*64))
                    );
                }
                fence_view_async_shared();
                NamedBarrier::arrive_and_wait(128, warpgroup_idx ? NamedBarriers::warpgroup1_sync : NamedBarriers::warpgroup0_sync);
                // S -> G
                if (s2g_pred) {
                    int g_tile_idx = warpgroup_idx*4 + tile_idx;
                    SM90_TMA_STORE_3D::copy(
                        &tma_params.tensor_map_O,
                        plan.q_o.o.data() + g_tile_idx*(B_H*64),
                        g_tile_idx*64,
                        q_h_idx*B_H,
                        s_q_idx
                    );
                }
            }
            cute::tma_store_arrive();
        };


        if (warpgroup_idx == 0) {
            // Warpgroup 0

            auto pipelined_wait_and_qkt_gemm_l = [&]() __attribute__((always_inline)) {
                plan.bar_k0_ready[0].wait(cur_bar_wait_phase);
                qkt_gemm_one_tile(Warpgroup0{}, 0, true);
                qkt_gemm_one_tile(Warpgroup0{}, 1, false);
                qkt_gemm_one_tile(Warpgroup0{}, 2, false);
                qkt_gemm_one_tile(Warpgroup0{}, 3, false);
                warpgroup_commit_batch();
            };

            auto pipelined_wait_and_qkt_gemm_r = [&]() __attribute__((always_inline)) {
                plan.bar_k0_ready[1].wait(cur_bar_wait_phase);
                qkt_gemm_one_tile(Warpgroup0{}, 4, false);
                qkt_gemm_one_tile(Warpgroup0{}, 5, false);
                qkt_gemm_one_tile(Warpgroup0{}, 6, false);
                qkt_gemm_one_tile(Warpgroup0{}, 7, false);
                qkt_gemm_one_tile(Warpgroup0{}, 8, false);
                warpgroup_commit_batch();
            };

            auto scale_rS = [&](float scales[2]) {
                CUTE_UNROLL
                for (int row = 0; row < 2; ++row) {
                    CUTE_UNROLL
                    for (int i = row*2; i < size(rP); i += 4) {
                        rS(i) = (bf16)(rP(i) * scales[row]);
                        rS(i+1) = (bf16)(rP(i+1) * scales[row]);
                    }
                }
            };

            auto rescale_rO = [&](float scales[2]) {
                CUTE_UNROLL
                for (int row = 0; row < 2; ++row) {
                    CUTE_UNROLL
                    for (int i = row*2; i < size(rO); i += 4) {
                        rO(i) *= scales[row];
                        rO(i+1) *= scales[row];
                    }
                    rL[row] *= scales[row];
                }
            };
            
            CUTE_NO_UNROLL
            for (int block_idx = 0; block_idx < num_topk_blocks; block_idx += 2) {
                Tensor sV0l = make_tensor(make_smem_ptr(plan.k[0].data()), SmemLayoutKTilesTransposed<4>{});
                Tensor sV1l = make_tensor(make_smem_ptr(plan.k[1].data()), SmemLayoutKTilesTransposed<4>{});

                if (block_idx == 0) {
                    // NOTE We put these code here to avoid register spilling
                    pipelined_wait_and_qkt_gemm_l();
                    pipelined_wait_and_qkt_gemm_r();
                    warpgroup_wait<0>();
                }
                
                // Online softmax, inform WG1
                mask_rP(Warpgroup0{});
                
                online_softmax_and_rescale_o(Warpgroup0{});
                NamedBarrier::arrive(256, NamedBarriers::wg0_bunch_0_ready);

                // Issue rO0 += rS0 @ sV0l
                gemm_rs(false, TiledMMA_PV_LocalP{}, rS, sV0l, rO, idx_in_warpgroup);
                warpgroup_commit_batch();

                // Mark V0L as free
                warpgroup_wait<0>();
                plan.bar_k0_free[0].arrive();

                // Wait for new sM, scale rS, save, inform WG1
                NamedBarrier::arrive_and_wait(256, NamedBarriers::wg1_bunch_0_ready);
                float new_rM[2], scale_factors[2];
                *(float2*)new_rM = plan.sM[idx_in_warpgroup/4];
                CUTE_UNROLL
                for (int i = 0; i < 2; ++i) {
                    scale_factors[i] = exp2f(rM[i] - new_rM[i]);
                    rM[i] = new_rM[i];
                }
                scale_rS(scale_factors);
                save_rS_to_sS(rS, sS0, idx_in_warpgroup);
                fence_view_async_shared();
                NamedBarrier::arrive(256, NamedBarriers::wg0_s0_ready);

                // Wait for sS1
                NamedBarrier::arrive_and_wait(256, NamedBarriers::wg1_s1_ready);

                // Rescale rO0, Issue rO0 += sS1 @ sV1L
                rescale_rO(scale_factors);
                gemm_ss(false, TiledMMA_PV_RemoteP{}, sS1, sV1l, rO, idx_in_warpgroup);
                warpgroup_commit_batch();

                cur_bar_wait_phase ^= 1;

                if (block_idx+2 < num_topk_blocks) {
                    // Launch the next QK^T GEMM
                    pipelined_wait_and_qkt_gemm_l();

                    // Mark V1L as free
                    warpgroup_wait<1>();
                    plan.bar_k1_free[0].arrive();
                    pipelined_wait_and_qkt_gemm_r();

                    // Wait for rP0 = sQ @ sK0
                    warpgroup_wait<0>();
                } else {
                    // Mark V1L as free
                    warpgroup_wait<0>();
                    plan.bar_k1_free[0].arrive();
                }
            }

            reduce_L();
            store_O();
        } else {
            // Warpgroup 1

            auto pipelined_wait_and_qkt_gemm = [&]() __attribute__((always_inline)) {
                plan.bar_k1_ready[1].wait(cur_bar_wait_phase);
                qkt_gemm_one_tile(Warpgroup1{}, 4, true);
                qkt_gemm_one_tile(Warpgroup1{}, 5, false);
                qkt_gemm_one_tile(Warpgroup1{}, 6, false);
                qkt_gemm_one_tile(Warpgroup1{}, 7, false);
                qkt_gemm_one_tile(Warpgroup1{}, 8, false);
                plan.bar_k1_ready[0].wait(cur_bar_wait_phase);
                qkt_gemm_one_tile(Warpgroup1{}, 0, false);
                qkt_gemm_one_tile(Warpgroup1{}, 1, false);
                qkt_gemm_one_tile(Warpgroup1{}, 2, false);
                qkt_gemm_one_tile(Warpgroup1{}, 3, false);
                warpgroup_commit_batch();
            };
            
            CUTE_NO_UNROLL
            for (int block_idx = 0; block_idx < num_topk_blocks; block_idx += 2) {
                Tensor sV0r = make_tensor(make_smem_ptr(plan.k[0].data()+64*256), SmemLayoutKTilesTransposed<4>{});
                Tensor sV1r = make_tensor(make_smem_ptr(plan.k[1].data()+64*256), SmemLayoutKTilesTransposed<4>{});

                // Issue rP1 = sQ @ sK1, and wait
                pipelined_wait_and_qkt_gemm();
                warpgroup_wait<0>();

                mask_rP(Warpgroup1{});

                // Wait for WG0 (for sM), online softmax, Notify WG0 (sM ready)
                NamedBarrier::arrive_and_wait(256, NamedBarriers::wg0_bunch_0_ready);
                online_softmax_and_rescale_o(Warpgroup1{});
                NamedBarrier::arrive(256, NamedBarriers::wg1_bunch_0_ready);


                // Issue rO1 += rS1 @ sV1R
                gemm_rs(false, TiledMMA_PV_LocalP{}, rS, sV1r, rO, idx_in_warpgroup);
                warpgroup_commit_batch();
                
                // Wait for WG0 (for sS0), Issue rO1 += rS0 @ sV0R
                save_rS_to_sS(rS, sS1, idx_in_warpgroup);   // Put it here is faster
                NamedBarrier::arrive_and_wait(256, NamedBarriers::wg0_s0_ready);
                gemm_ss(false, TiledMMA_PV_RemoteP{}, sS0, sV0r, rO, idx_in_warpgroup);
                warpgroup_commit_batch();
                
                // Save rS1, inform WG0
                fence_view_async_shared();
                NamedBarrier::arrive(256, NamedBarriers::wg1_s1_ready);

                // Wait for GEMM, and inform that sV1R is free
                warpgroup_wait<1>();
                plan.bar_k1_free[1].arrive();

                // Wait for GEMM, and inform that sV0R is free
                warpgroup_wait<0>();
                plan.bar_k0_free[1].arrive();

                cur_bar_wait_phase ^= 1;
            }

            reduce_L();
            store_O();

            // Save lse
            if (idx_in_warpgroup%4 == 0) {
                for (int row = 0; row < 2; ++row) {
                    int real_row = get_AorC_row_idx(row, idx_in_warpgroup);
                    bool is_no_valid_tokens = rL[row] == 0.0f;
                    plan.final_max_logits[real_row] = is_no_valid_tokens ? -INFINITY : rM[row];
                    plan.final_lse[real_row] = is_no_valid_tokens ? -INFINITY : log2f(rL[row]) + rM[row];
                }
                fence_view_async_shared();
            }

            NamedBarrier::arrive_and_wait(128, NamedBarriers::warpgroup1_sync);
            if (idx_in_warpgroup == 0) {
                int g_offset = s_q_idx*params.h_q + q_h_idx*B_H;
                SM90_BULK_COPY_S2G::copy(plan.final_max_logits, params.max_logits + g_offset, B_H*sizeof(float));
                SM90_BULK_COPY_S2G::copy(plan.final_lse, params.lse + g_offset, B_H*sizeof(float));
                cute::tma_store_arrive();
            }
        }
    } else {
        // Producer warpgroup
        cutlass::arch::warpgroup_reg_dealloc<72>();

        constexpr int GROUP_SIZE = 8, NUM_GROUPS = 128/GROUP_SIZE;
        constexpr int NUM_ROWS_PER_GROUP = B_TOPK / NUM_GROUPS;
        int idx_in_group = idx_in_warpgroup % GROUP_SIZE;
        int group_idx = idx_in_warpgroup / GROUP_SIZE;
        int* gIndices = params.indices + s_q_idx*params.topk;   // [topk]

        bf16* my_sKV_base = &(make_tensor(make_smem_ptr(plan.k[0].data()), SmemLayoutKTiles<1>{})(group_idx, idx_in_group*8));
        bf16* my_gKV_base = params.kv + idx_in_group*8;
        
        int64_t token_indices[2][NUM_ROWS_PER_GROUP];
        bool is_token_valid[2][NUM_ROWS_PER_GROUP];
        auto load_token_indices = [&](int block_idx) {
            CUTE_UNROLL
            for (int buf_idx = 0; buf_idx < 2; ++buf_idx) {
                CUTE_UNROLL
                for (int local_row = 0; local_row < NUM_ROWS_PER_GROUP; ++local_row) {
                    int offs = (block_idx+buf_idx)*B_TOPK + local_row*NUM_GROUPS + group_idx;
                    int t = __ldg(gIndices + offs);
                    token_indices[buf_idx][local_row] = t*(int64_t)params.stride_kv_s_kv;   // We mult it with params.stride_kv_s_kv here since it's faster
                    is_token_valid[buf_idx][local_row] = t >= 0 && t < params.s_kv;
                }
            }
        };
        
        int64_t cache_policy = createpolicy_evict_last();
        auto copy_tiles = [&](int block_idx, int buf_idx, int tile_start, int tile_end) {
            // Copy some K/V tiles from global memory to shared memory
            // A tile has a shape of 64 (B_TOPK) x 64
            // `buf_idx` is the index of the shared memory buffer, 0 or 1
            // `tile_idx` is the index of the tile to load, from 0 to D_K/64-1 = 8
            CUTE_UNROLL
            for (int local_row = 0; local_row < NUM_ROWS_PER_GROUP; ++local_row) {
                int64_t token_index = token_indices[buf_idx][local_row];
                CUTE_UNROLL
                for (int tile_idx = tile_start; tile_idx < tile_end; ++tile_idx) {
                    cp_async_cacheglobal_l2_prefetch_256B(
                        my_gKV_base + token_index + tile_idx*64,
                        my_sKV_base + (buf_idx*B_TOPK*D_K + tile_idx*(B_TOPK*64) + local_row*NUM_GROUPS*64),
                        is_token_valid[buf_idx][local_row],
                        cache_policy
                    );
                }
            }
        };

        auto commit_to_mbar = [&](transac_bar_t &bar) {
            cutlass::arch::cpasync_barrier_arrive_noinc((uint64_t*)(&bar));
        };

        int cur_bar_wait_phase = 1;

        CUTE_NO_UNROLL
        for (int block_idx = 0; block_idx < num_topk_blocks; block_idx += 2) {
            load_token_indices(block_idx);

            // V0L
            plan.bar_k0_free[0].wait(cur_bar_wait_phase);
            copy_tiles(block_idx+0, 0, 0, 4);
            commit_to_mbar(plan.bar_k0_ready[0]);

            // V1R
            plan.bar_k1_free[1].wait(cur_bar_wait_phase);
            copy_tiles(block_idx+1, 1, 4, 9);
            commit_to_mbar(plan.bar_k1_ready[1]);
            
            // V0R
            plan.bar_k0_free[1].wait(cur_bar_wait_phase);
            copy_tiles(block_idx+0, 0, 4, 9);
            commit_to_mbar(plan.bar_k0_ready[1]);

            // V1L
            plan.bar_k1_free[0].wait(cur_bar_wait_phase);
            copy_tiles(block_idx+1, 1, 0, 4);
            commit_to_mbar(plan.bar_k1_ready[0]);

            // Valid mask
            // NOTE V1R's finish implies maskings of the last round have finished
            if (idx_in_group == 0) {
                CUTE_UNROLL
                for (int buf_idx = 0; buf_idx < 2; ++buf_idx)
                    CUTE_UNROLL
                    for (int local_row = 0; local_row < NUM_ROWS_PER_GROUP; ++local_row)
                        plan.is_kv_valid[buf_idx][local_row*NUM_GROUPS+group_idx] = is_token_valid[buf_idx][local_row];
                plan.bar_is_kv_valid_ready.arrive();
            }

            cur_bar_wait_phase ^= 1;
        }
    }
#else
    if (cute::thread0()) {
        CUTE_INVALID_CONTROL_PATH("This kernel only supports sm90");
    }
#endif
}


void run_fwd_kernel(const SparsePrefillParams& params) {
    FLASH_ASSERT(params.h_kv == 1);
    FLASH_ASSERT(params.topk % (2*B_TOPK) == 0);   // To save some boundry checkings
    FLASH_ASSERT(params.topk > 0);
    FLASH_ASSERT(params.h_q % B_H == 0);

    auto shape_Q = make_shape(params.h_q, params.d_qk, params.s_q);
    auto tma_Q = cute::make_tma_copy(
        SM90_TMA_LOAD{},
        make_tensor(
            make_gmem_ptr((bf16*)params.q),
            make_layout(
                shape_Q,
                make_stride(params.stride_q_h_q, _1{}, params.stride_q_s_q)
            )
        ),
        SmemLayoutQ{}
    );

    CUtensorMap tensor_map_O;
    {
        uint64_t size[3] = {D_V, (unsigned long)params.h_q, (unsigned long)params.s_q};
        uint64_t stride[2] = {D_V*sizeof(bf16), D_V*params.h_q*sizeof(bf16)};
        uint32_t box_size[3] = {64, B_H, 1};
        uint32_t elem_stride[3] = {1, 1, 1};
        hipError_t res = CUTLASS_CUDA_DRIVER_WRAPPER_CALL(cuTensorMapEncodeTiled)(
            &tensor_map_O,
            CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_BFLOAT16,
            3,
            params.out,
            size,
            stride,
            box_size,
            elem_stride,
            CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE,
            CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_128B,
            CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE,
            CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE
        );
        FLASH_ASSERT(res == hipError_t::hipSuccess);
    }

    TmaParams<
        decltype(shape_Q), decltype(tma_Q)
    > tma_params = {
        shape_Q, tma_Q,
        tensor_map_O
    };
    auto kernel = &sparse_attn_fwd_kernel<decltype(tma_params)>;

    constexpr size_t smem_size = sizeof(SharedMemoryPlan);
    CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));

    cutlass::ClusterLaunchParams launch_params = {
        dim3((params.h_q/B_H)*params.s_q, 1, 1),    // NOTE We put s_q on the first dim since it can be larger than 65536 (the maximum size of griddim.y and griddim.z)
        dim3(NUM_THREADS, 1, 1),
        dim3(1, 1, 1),
        smem_size,
        params.stream
    }; 
    cutlass::launch_kernel_on_cluster(
        launch_params, (void*)kernel, params, tma_params
    );
    CHECK_CUDA_KERNEL_LAUNCH();
}

}

#else // !SM90+ architecture

namespace sm90 {

void run_fwd_kernel(const SparsePrefillParams& params) {
    throw std::runtime_error("FlashMLA sparse prefill kernel requires SM90+ architecture. This build was compiled without SM90 support.");
}

}

#endif // SM90+ architecture check
