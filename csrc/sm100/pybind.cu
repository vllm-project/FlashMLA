#include <torch/python.h>

void FMHACutlassSM100FwdRun(at::Tensor workspace_buffer, at::Tensor q, at::Tensor k, at::Tensor v,
                            at::Tensor cumulative_seqlen_q, at::Tensor cumulative_seqlen_kv,
                            at::Tensor o, at::Tensor lse,
                            int mask_mode_code, float softmax_scale, int max_seqlen_q, int max_seqlen_kv, bool is_varlen);

void FMHACutlassSM100BwdRun(at::Tensor workspace_buffer, at::Tensor d_o, at::Tensor q, at::Tensor k,
                            at::Tensor v, at::Tensor o, at::Tensor lse,
                            at::Tensor cumulative_seqlen_q, at::Tensor cumulative_seqlen_kv,
                            at::Tensor dq, at::Tensor dk, at::Tensor dv,
                            int mask_mode_code, float softmax_scale, int max_seqlen_q, int max_seqlen_kv, bool is_varlen);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fwd", &FMHACutlassSM100FwdRun);
    m.def("bwd", &FMHACutlassSM100BwdRun);
}
