// SM100-specific file - only compile for SM100+ architectures
#include "interface.h"
#include <stdexcept>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000) || !defined(__CUDA_ARCH__)

#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_bf16.h>

#include "common/mask.cuh"
#include "common/utils.hpp"

#include "fmha_cutlass_fwd_sm100.cuh"

template <class Mask, class Varlen, class Element, class ElementOut, class Mla>
void call_run_fmha_fwd([[maybe_unused]] Mask mask, [[maybe_unused]] Varlen is_varlen,
                       [[maybe_unused]] Element in, [[maybe_unused]] ElementOut out,
                       [[maybe_unused]] Mla mla, at::Tensor workspace_buffer, at::Tensor q,
                       at::Tensor k, at::Tensor v, at::Tensor cumulative_seqlen_q,
                       at::Tensor cumulative_seqlen_kv, at::Tensor o, at::Tensor lse,
                       float softmax_scale, int max_seqlen_q, int max_seqlen_kv) {
  static constexpr bool IsVarlen = std::is_same_v<Varlen, true_type>;
  static constexpr bool IsMla = std::is_same_v<Mla, true_type>;
  static constexpr bool IsCausalMask = std::is_same_v<Mask, CausalMask<false>>;
  using Option =
      std::conditional_t<IsCausalMask || (IsVarlen), Option<Tag::kIsPersistent, false_type>,
                         Option<Tag::kIsPersistent, true_type>>;

  run_fmha_fwd<Element, ElementOut, IsVarlen, IsMla, Mask, Option>(
      workspace_buffer, q, k, v, cumulative_seqlen_q, cumulative_seqlen_kv, o, lse,
      softmax_scale, max_seqlen_q, max_seqlen_kv);
}

void FMHACutlassSM100FwdRun(at::Tensor workspace_buffer, at::Tensor q, at::Tensor k,
                            at::Tensor v, at::Tensor cumulative_seqlen_q,
                            at::Tensor cumulative_seqlen_kv, at::Tensor o, at::Tensor lse,
                            int mask_mode_code, float sm_scale, int max_seqlen_q,
                            int max_seqlen_kv, bool is_varlen) {
  const c10::cuda::OptionalCUDAGuard device_guard(q.device());
  CHECK(q.scalar_type() == k.scalar_type());
  auto scalar_type_in = q.scalar_type();
  auto scalar_type_out = o.scalar_type();
  int head_dim_qk = q.size(-1);
  int head_dim_vo = v.size(-1);
  MaskMode mask_mode = static_cast<MaskMode>(mask_mode_code);

  if (scalar_type_in == at::ScalarType::BFloat16 &&
      scalar_type_out == at::ScalarType::BFloat16) {
    using Element = cutlass::bfloat16_t;
    using ElementOut = cutlass::bfloat16_t;

    auto apply_config = [&](auto fn) {
      if (mask_mode == MaskMode::kCausal) {
        if (is_varlen) {
          fn(CausalMask<false>{}, cute::true_type{}, Element{}, ElementOut{});
        } else {
          fn(CausalMask<false>{}, cute::false_type{}, Element{}, ElementOut{});
        }
      } else {
        if (is_varlen) {
          fn(ResidualMask{}, cute::true_type{}, Element{}, ElementOut{});
        } else {
          fn(ResidualMask{}, cute::false_type{}, Element{}, ElementOut{});
        }
      }
    };

    apply_config([&](auto mask, auto varlen, auto in, auto out) {
      if (head_dim_qk == 192 && head_dim_vo == 128) {
        call_run_fmha_fwd(mask, varlen, in, out, true_type{}, workspace_buffer, q, k, v,
                          cumulative_seqlen_q, cumulative_seqlen_kv, o, lse, sm_scale,
                          max_seqlen_q, max_seqlen_kv);
      } else if (head_dim_qk == 128 && head_dim_vo == 128) {
        call_run_fmha_fwd(mask, varlen, in, out, false_type{}, workspace_buffer, q, k, v,
                          cumulative_seqlen_q, cumulative_seqlen_kv, o, lse, sm_scale,
                          max_seqlen_q, max_seqlen_kv);
      } else {
        std::cout << "No kernel instantiated for head_dim_qk=" << head_dim_qk
                  << " head_dim_vo=" << head_dim_vo << std::endl;
      }
    });

  } else {
    FLASH_MLA_ASSERT(false);
  }
}

#else // !SM100+ architecture

void FMHACutlassSM100FwdRun(at::Tensor workspace_buffer, at::Tensor q, at::Tensor k,
                           at::Tensor v, at::Tensor cumulative_seqlen_q,
                           at::Tensor cumulative_seqlen_kv, at::Tensor o, at::Tensor lse,
                           int mask_mode_code, float sm_scale, int max_seqlen_q,
                           int max_seqlen_kv, bool is_varlen) {
    throw std::runtime_error("FlashMLA dense prefill requires SM100+ architecture. This build was compiled without SM100 support.");
}

void FMHACutlassSM100BwdRun(at::Tensor workspace_buffer, at::Tensor d_o, at::Tensor q, at::Tensor k,
                           at::Tensor v, at::Tensor o, at::Tensor lse,
                           at::Tensor cumulative_seqlen_q, at::Tensor cumulative_seqlen_kv,
                           at::Tensor dq, at::Tensor dk, at::Tensor dv,
                           int mask_mode_code, float softmax_scale, int max_seqlen_q, int max_seqlen_kv, bool is_varlen) {
    throw std::runtime_error("FlashMLA dense prefill backward requires SM100+ architecture. This build was compiled without SM100 support.");
}

#endif // SM100+ architecture check
